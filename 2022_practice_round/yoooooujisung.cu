#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}
using namespace std;

int C, L;
string good_raw[10001][5];
string bad_raw[10001][5];

vector<int> good[10001];
vector<int> bad[10001];

set<string> ingredients_raw;

int graph[10001][10001];

__global__ void eval_cu(int *answer_cu, int *answer, int answer_len, int *good_cu, int *bad_cu, int *good_len, int *bad_len, int C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < C)
    {
        int clients = 1;
        for (int j = 0; j < good_len[i]; j++)
        {
            bool check = true;
            for (int k = 0; k < answer_len; k++)
            {
                if (answer[k] == good_cu[i * 10 + j])
                    check = false;
            }
            if (check)
                clients = 0;
        }
        for (int j = 0; j < bad_len[i]; j++)
        {
            bool check = false;
            for (int k = 0; k < answer_len; k++)
            {
                if (answer[k] == bad_cu[i * 10 + j])
                    check = true;
            }
            if (check)
                clients = 0;
        }
        answer_cu[i] = clients;
    }
}

int eval(set<int> answer)
{
    int cnt = 0;
    for (int i = 0; i < C; i++)
    {
        bool clients = true;
        for (int j = 0; j < good[i].size(); j++)
            if (answer.find(good[i][j]) == answer.end())
                clients = false;
        for (int j = 0; j < bad[i].size(); j++)
            if (answer.find(bad[i][j]) != answer.end())
                clients = false;
        if (clients)
            cnt++;
    }
    return cnt;
}

int main(int argc, char const *argv[])
{
    ifstream fin;
    fin.open(argv[1]);

    ofstream fout;
    fout.open(argv[2]);

    fin >> C;

    int *good_cu;
    int *bad_cu;
    int *good_len;
    int *bad_len;

    hipMallocManaged(&good_cu, 10 * C * sizeof(int));
    hipMallocManaged(&bad_cu, 10 * C * sizeof(int));
    hipMallocManaged(&good_len, C * sizeof(int));
    hipMallocManaged(&bad_len, C * sizeof(int));

    for (int i = 0; i < C; i++)
    {
        int num;
        fin >> num;
        good_len[i] = num;
        for (int j = 0; j < num; j++)
        {
            fin >> good_raw[i][j];
            ingredients_raw.insert(good_raw[i][j]);
        }
        fin >> num;
        bad_len[i] = num;
        for (int j = 0; j < num; j++)
        {
            fin >> bad_raw[i][j];
            ingredients_raw.insert(bad_raw[i][j]);
        }
    }
    vector<string> ingredients(ingredients_raw.begin(), ingredients_raw.end());

    for (int i = 0; i < C; i++)
    {
        for (int j = 0; j < good_len[i]; j++)
        {
            vector<string>::iterator iter = find(ingredients.begin(), ingredients.end(), good_raw[i][j]);
            int index = distance(ingredients.begin(), iter);
            good[i].push_back(index);
            good_cu[i * 10 + j] = index;
        }
        for (int j = 0; j < bad_len[i]; j++)
        {
            vector<string>::iterator iter = find(ingredients.begin(), ingredients.end(), bad_raw[i][j]);
            int index = distance(ingredients.begin(), iter);
            bad[i].push_back(index);
            bad_cu[i * 10 + j] = index;
        }
    }

    int edge = 0;
    for (int i = 0; i < C; i++)
    {
        for (int j = 0; j < C; j++)
        {
            for (int k = 0; k < good[i].size(); k++)
            {
                if (find(bad[j].begin(), bad[j].end(), good[i][k]) != bad[j].end())
                {
                    graph[i][j] = 1;
                    edge++;
                }
            }
            for (int k = 0; k < bad[i].size(); k++)
            {
                if (find(good[j].begin(), good[j].end(), bad[i][k]) != good[j].end())
                {
                    graph[i][j] = 1;
                    edge++;
                }
            }
        }
    }
    cout << "number of edge : " << edge << endl;

    // print graph
    // for (int i = 0; i < C; i++)
    // {
    //     for (int j = 0; j < C; j++)
    //     {
    //         cout << graph[i][j] << " ";
    //     }
    //     cout << endl;
    // }

    // degree initialization
    int check[10001] = {
        0,
    };
    int degree[10001] = {
        0,
    };

    for (int i = 0; i < C; i++)
    {
        for (int j = 0; j < C; j++)
        {
            degree[i] += graph[i][j];
        }
    }
    // cout << "degree initialization" << endl;

    // remove the vertex of the lowest degree in graph
    int cnt = 0;
    set<int> answer;
    set<int> vertex;

    while (true)
    {
        int min_degree = C;
        int min_degree_vertex = -1;
        for (int i = 0; i < C; i++)
        {
            if (check[i] == 1)
                continue;
            if (degree[i] < min_degree)
            {
                min_degree = degree[i];
                min_degree_vertex = i;
            }
        }
        if (min_degree == C)
            break;

        // cout << "D:" << min_degree << " " << min_degree_vertex << " ";
        vertex.insert(min_degree_vertex);
        for (int i = 0; i < good[min_degree_vertex].size(); i++)
        {
            answer.insert(good[min_degree_vertex][i]);
        }
        check[min_degree_vertex] = 1;
        for (int i = 0; i < C; i++)
        {
            if (check[i] == 0 && graph[min_degree_vertex][i] == 1)
            {
                check[i] = 1;
                for (int j = 0; j < C; j++)
                {
                    if (check[j] == 0 && graph[i][j] == 1)
                        degree[j]--;
                }
            }
        }
        cnt++;
    }
    cout << cnt << endl;

    int real = eval(answer);

    int *answer_copy;
    int *d_eval;
    hipMallocManaged(&answer_copy, C * sizeof(int));
    hipMallocManaged(&d_eval, C * sizeof(int));

    for (int cnt = 0; cnt < 10; cnt++)
    {

        float start = clock();
        for (int i = 0; i < 30; i++)
        {
            set<int> copy = answer;
            for (auto ans : copy)
            {
                answer.erase(ans);

                // CUDA
                int j = 0;
                for (auto it = answer.begin(); it != answer.end(); ++it)
                    answer_copy[j++] = *it;

                eval_cu<<<512, 32>>>(d_eval, answer_copy, int(answer.size()), good_cu, bad_cu, good_len, bad_len, C);
                gpuErrchk(hipDeviceSynchronize());

                int score = 0;
                for (int j = 0; j < C; j++)
                {
                    if (d_eval[j] == 1)
                        score++;
                }

                // CPU
                // int score = eval(answer);

                if (score <= real)
                    answer.insert(ans);
                else
                {
                    real = score;
                    break;
                }
            }
        }
        float end = clock();
        float time = (end - start) / CLOCKS_PER_SEC;

        printf("time: %f\n", time);
        cout << "[+]: " << eval(answer) << endl;

        for (int i = 0; i < 30; i++)
        {
            for (int igd = 0; igd < ingredients.size(); igd++)
            {
                if (answer.find(igd) != answer.end())
                    continue;
                answer.insert(igd);

                // CUDA
                int j = 0;
                for (auto it = answer.begin(); it != answer.end(); ++it)
                    answer_copy[j++] = *it;

                eval_cu<<<512, 32>>>(d_eval, answer_copy, int(answer.size()), good_cu, bad_cu, good_len, bad_len, C);
                gpuErrchk(hipDeviceSynchronize());

                int score = 0;
                for (int j = 0; j < C; j++)
                {
                    if (d_eval[j] == 1)
                        score++;
                }

                // CPU
                // int score = eval(answer);

                if (score < real)
                {
                    answer.erase(igd);
                }
                else
                {
                    real = score;
                    break;
                }
            }
        }
        cout << "[+]: " << eval(answer) << endl;
    }
    fout << answer.size() << " ";

    for (auto a : answer)
        fout << ingredients[a] << " ";

    hipFree(good_cu);
    hipFree(bad_cu);
    hipFree(good_len);
    hipFree(bad_len);
    hipFree(answer_copy);
    hipFree(d_eval);
}